// Author Bryan Boule : bryanboule@gmail.com


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

// Chose list length
//define taille_a 10
//define taille_b 10

__global__ void parallel_merge_path(int *A, int *B, int *M, int taille_a, int taille_b){
	
	// Define variables
	int i = threadIdx.x+blockIdx.x*blockDim.x;
	int atop;
	int btop;
	int abottom;
	int offset;
	int ai;
	int bi;
	if (i > taille_a){
		atop = taille_a;
		btop = i - taille_a;
	} else {
		atop = i;
		btop = 0;
	}
	abottom = btop;
	while (1){
		offset = abs(atop - abottom)/2;
		ai = atop - offset;
		bi = btop + offset;
		if (ai >= 0 && bi <= taille_b && (ai == taille_a || bi == 0 || A[ai] > B[bi-1])){
			if (bi == taille_b || ai == 0 || A[ai-1] <= B[bi]){
				if ((ai < taille_a) && ((bi == taille_b) || (A[ai] <= B[bi]))){
					M[i] = A[ai];
				} else {
					M[i] = B[bi];
				}
				break;
			} else {
				atop = ai - 1;
				btop = bi + 1;
			}
		} else {
			abottom = ai + 1;
		}
	}
}

int main(){
	
	// Define variables
	int *a_array;
	int *b_array;
	int *m_array;
	int *aGPU, *bGPU, *mGPU;
	hipEvent_t start, stop;

	int taille_a = 1027;
	int taille_b = 2053;
	
	float TimeVar;


	// Allocate memory
	a_array = (int *)malloc(sizeof(int)*taille_a);
    b_array = (int *)malloc(sizeof(int)*taille_b);
	m_array = (int *)malloc(sizeof(int)*(taille_a + taille_b));

	// Test on sorted separated list generated with increasing function
	for(int i=0; i<taille_a; i++){
		a_array[i] = 3*i+2;
	}
	for(int j=0; j<taille_b; j++){
		b_array[j] = 2*j+4;
	}
	
	for(int j=0; j<(taille_a+taille_b); j++){
		m_array[j] = 0;
	}

	hipMalloc(&aGPU, taille_a*sizeof(int));
	hipMalloc(&bGPU, taille_b*sizeof(int));
	hipMalloc(&mGPU, (taille_a+taille_b)*sizeof(int));
	
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipMemcpy(aGPU,a_array,taille_a*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(bGPU,b_array,taille_b*sizeof(int), hipMemcpyHostToDevice);

	hipEventRecord(start,0);
	parallel_merge_path<<<(1+(taille_a+taille_b)/1024), (taille_a+taille_b)/(1+(taille_a + taille_b)/1024) + ((taille_a + taille_b)%(1+(taille_a+taille_b)/1024))>>>(aGPU,bGPU,mGPU,taille_a, taille_b);
	hipDeviceSynchronize();
	hipMemcpy(m_array, mGPU,(taille_a+taille_b)*sizeof(int), hipMemcpyDeviceToHost);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&TimeVar, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	printf("Temps d'éxécution : %f\n", TimeVar);
	
	hipFree(aGPU);
	hipFree(bGPU);
	hipFree(mGPU);
	
	// Display result
    for (int i=0; i <(taille_a+taille_b); i++){
        printf("%d  ", m_array[i]);
	}
	printf("\n");
	
	// Free memory
	free(a_array);
	free(b_array);
	free(m_array);

	return 0;
}
